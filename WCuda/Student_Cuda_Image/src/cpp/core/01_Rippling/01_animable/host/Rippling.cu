#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Rippling.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Rippling_Cuda_RGBA_uchar4")
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Rippling::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Rippling::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

