#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RipplingMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;


    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID; // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

	ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t); // update ptrTabPixels[s]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

