#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "RayTracingMath.h"
#include "length_cm.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

// Déclaration Constante globale
__constant__ Sphere TAB_CM[NB_SPHERE];
/**
 * call once by the host
 */
__host__ void uploadToCM(Sphere* tabValue)
    {
    size_t size = NB_SPHERE * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabValue, size, offset, hipMemcpyHostToDevice));
    }

__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere)
    {
    RayTracingMath rayTracingMath = RayTracingMath(t,nbSphere, ptrDevTabSphere);

        const int TID = Indice2D::tid();
        const int NB_THREAD = Indice2D::nbThread();
        const int WH = w * h;


        int i;	// in [0,h[
        int j; 	// in [0,w[

        int s = TID; // in [0,...
        while (s < WH)
    	{
    	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

    	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j); // update ptrTabPixels[s]

    	s += NB_THREAD;
    	}
    }

__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere)
    {
    //global memory
    work(ptrDevPixels,w,h,t,ptrDevTabSphere,nbSphere);
    }

__global__ void rayTracingToCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere)
    {
    //constant memory
    work(ptrDevPixels,w,h,t,TAB_CM,nbSphere);
    }

__device__ void copyGMtoSM(Sphere* tabGM, Sphere* tabSM, int n)
    {
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    const int TID_LOCAL = Indice2D::tidBlock();
    int s = TID_LOCAL;
    while(s<n)
	{
	tabSM[s]=tabGM[s];
	s+=NB_THREAD_LOCAL;
	}
    }

__global__ void rayTracingToSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere, int nbSphere)
    {
    //shared mem
    __shared__ extern Sphere tabSM[];
    copyGMtoSM(ptrDevTabSphere,tabSM,nbSphere);
    __syncthreads();
    work(ptrDevPixels,w,h,t,tabSM,nbSphere);
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/



/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

