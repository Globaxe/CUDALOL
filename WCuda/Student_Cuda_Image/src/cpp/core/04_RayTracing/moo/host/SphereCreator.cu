#include "hip/hip_runtime.h"
#include "SphereCreator.h"
#include "AleaTools.h"
#include <iostream>
#include <assert.h>
/*--------------------------------------*\
|*
 Constructeur
 *|
 \*-------------------------------------*/
SphereCreator::SphereCreator(int nbSpheres, int w, int h, int bord)
    {
    assert(nbSpheres >= 4);
// Inputs
    this->nbSpheres = nbSpheres;
    this->w = w;
    this->h = h;
    this->bord = bord;
// Outputs
    this->tabSphere = new Sphere[nbSpheres];
// required default constructeur of type : sphere(void)
    createSphereAll();
    }
SphereCreator::~SphereCreator()
    {
    delete[] tabSphere;
    }
/*--------------------------------------*\
|*
 Methode
 *|
 \*-------------------------------------*/
Sphere* SphereCreator::getTabSphere()
    {
    return tabSphere;
    }
/*--------------------------------------*\
|*
 Private
 *|
 \*-------------------------------------*/
void SphereCreator::createSphereAll(void)
    {
    createSphereBack();
    createSphereFront();
    createSphereAleatoire();
    }
/**
 * La premiere, centrée, très grande, au fond
 */
void SphereCreator::createSphereBack(void)
    {
    float3 centre;
    centre.x = h / 2;
    centre.y = w / 2;
    centre.z = ZMAX * 2; // plus grand que ZMAX use dans alea
    float rayon = std::min(w / 2, h / 2);
    float hue01 = 1;
    tabSphere[0] = Sphere(centre, rayon, hue01);
    }
/**
 * Les dernieres, centrées, très petite, très proche
 */
void SphereCreator::createSphereFront(void)
    {
    float hue01 = 1;
    float rayon = std::min(w / 10, h / 10); // disons
    float z = ZMIN - (rayon + rayonMax());
    float dhue = 0.2;
    float dz = 40;
    float drayon = dz / 2;
    for (int i = nbSpheres - 3; i < nbSpheres; i++)
	{
	float3 centre;
	z -= dz;
	rayon -= drayon;
	hue01 -= dhue;
	centre.x = h / 2;
	centre.y = w / 2;
	centre.z = z;
	tabSphere[i] = Sphere(centre, rayon, hue01);
	}
    }
/**
 * Toutes les autres, aleatoires
 */
void SphereCreator::createSphereAleatoire(void)
    {
    AleaTools aleaTools = AleaTools();
    for (int i = 1; i < nbSpheres - 3; i++)
	{
	float3 centre;
	centre.x = aleaTools.uniformeAB(0 + bord, h - bord);
	centre.y = aleaTools.uniformeAB(0 + bord, w - bord);
	centre.z = aleaTools.uniformeAB(ZMIN, ZMAX);
	float rayon = aleaTools.uniformeAB(20, rayonMax());
	float hue01 = aleaTools.uniforme01();
	tabSphere[i] = Sphere(centre, rayon, hue01);
// Warning : sphere stack copier dans sphere heap,
// ok car pas ptr dans Sphere
	}
    }
int SphereCreator::rayonMax()
    {
    return w / 10;
    }
