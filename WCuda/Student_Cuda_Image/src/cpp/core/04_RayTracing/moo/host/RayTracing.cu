#include "hip/hip_runtime.h"
#include "RayTracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>
#include "length_cm.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h,float t, Sphere* ptrDevTabSphere, int nbSphere);
extern __global__ void rayTracingToCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSphere);
extern void uploadToCM(Sphere* tabValue);


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/



/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, int nbSphere, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
    {
    SphereCreator sphereCreator(nbSphere,w,h);
    Sphere* ptrTabSphere=sphereCreator.getTabSphere();
    this->sizeOctet = nbSphere*sizeof(Sphere);
    this->nbSphere = nbSphere;

    // global memory et shared mem
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevTabSphere, sizeOctet);
	    }

	// MM (copy Host->Device)
	    {
	    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
	    }
	}

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable


    //constant memory
    //uploadToCM(ptrTabSphere);
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("RayTracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    //global mem
    //rayTracing<<<dg,db>>>(ptrDevPixels,w,h,t, ptrDevTabSphere,this->nbSphere);
    //constant mem
    //rayTracingToCM<<<dg,db>>>(ptrDevPixels,w,h,t,this->nbSphere);
    //shared mem
    rayTracing<<<dg,db,this->sizeOctet>>>(ptrDevPixels,w,h,t, ptrDevTabSphere,this->nbSphere);

    Device::lastCudaError("RayTracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

