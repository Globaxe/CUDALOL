#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"

#include "Slice.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void slice(float* ptrDevTabGM,int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int n) :
	n(n)
    {
    this->sizeOctet = n * sizeof(float); // octet

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrTab, sizeOctet);
	    Device::malloc(&ptrTabGM, sizeOctet);
	    }

	Device::lastCudaError("Slice MM (end allocation)"); // temp debug, facultatif
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(ptrTab);
	Device::free(ptrTabGM);

	Device::lastCudaError("Slice MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    Device::lastCudaError("slice (before)"); // temp debug
    slice<<<dg,db>>>(ptrTabGM, n); // assynchrone
    Device::lastCudaError("slice (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	//Device::memcpyDToH(ptrW, ptrDevW, sizeOctet); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
