#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid, float* ptrV1, float* ptrV2, float* ptrW, int n) :
	ptrV1(ptrV1), ptrV2(ptrV2), ptrW(ptrW), n(n)
    {
    this->sizeOctet = n * sizeof(float); // octet

    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevV1, sizeOctet);
	    Device::malloc(&ptrDevV2, sizeOctet);
	    Device::malloc(&ptrDevW, sizeOctet);
	    }

	// MM (copy Host->Device)
	    {
	    Device::memcpyHToD(ptrDevV1, ptrV1, sizeOctet);
	    Device::memcpyHToD(ptrDevV2, ptrV2, sizeOctet);
	    }

	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug, facultatif
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevV1);
	Device::free(ptrDevV2);
	Device::free(ptrDevW);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddVector::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug
    addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	Device::memcpyDToH(ptrW, ptrDevW, sizeOctet); // barriere synchronisation implicite
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
