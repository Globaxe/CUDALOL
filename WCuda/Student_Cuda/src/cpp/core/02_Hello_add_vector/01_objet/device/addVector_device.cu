#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {
    secondaire(ptrDevV1, ptrDevV2, ptrDevW, n); // pas necessaire, just for fun
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    // Debug, facultatif
    //if (TID == 0)
    //{
    //printf("Coucou from device tid = %d", TID); //required   Device::synchronize(); after the call of kernel
    //}

    // pattern 1-1
    if (TID < n) // facultatif mais plus sûr
	ptrDevW[TID] = ptrDevV1[TID] + ptrDevV2[TID];

    // pattern entrelacement
//    int s = TID;
//    while (s < n)
//	{
//	ptrDevW[s] = ptrDevV1[s] + ptrDevV2[s];
//	s += NB_THREAD;
//	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

