#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"

#include "MonteCarloMultiGPU.h"
#include "MonteCarlo.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void monteCarlo(float* ptrDevTabGM, uint n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

MonteCarloMultiGPU::MonteCarloMultiGPU(const Grid& grid, uint n, float m)
    {
    this->grid = grid;
    this->n = n;
    this->m = m;
    this->pi = 0;
    }

MonteCarloMultiGPU::~MonteCarloMultiGPU(void)
    {

    }
/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MonteCarloMultiGPU::run()
    {
    uint n0 = 0;

#pragma omp parallel for reduction(+:n0)
    for (int deviceId = 0; deviceId < Device::getDeviceCount(); deviceId++)
	{
	Device::setDevice(deviceId);
	MonteCarlo montecarlo(grid, n/Device::getDeviceCount(), m);
	montecarlo.run();

	n0 += montecarlo.getN0();
	}

    pi = 4.f * n0 / n ;
    }

float MonteCarloMultiGPU::getPI()
    {
    return pi;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
