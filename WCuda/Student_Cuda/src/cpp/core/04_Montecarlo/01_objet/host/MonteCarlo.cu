#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"

#include "MonteCarlo.h"
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void monteCarlo(uint* ptrDevTabGM, hiprandState* ptrTabDevGeneratorGM, uint n);

extern __global__ void createGenerator(hiprandState* tabDevGeneratorGM,int deviceId);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

MonteCarlo::MonteCarlo(const Grid& grid, uint n, float m) :
	n(n)
    {
    this->sizeOctetGM = sizeof(uint); // octet
    this->sizeOctetSM = grid.db.x * sizeof(uint); // octet

    this->m = m;
    this->pi = 0;
    this->result = 0;
    this->n0 = 0;
    this->nbThread = grid.threadCounts();
    this->nbThread = this->n / grid.threadCounts();
    this->n = this->nbThread * grid.threadCounts();
    size_t sizeOctetGeneratorGM = grid.threadCounts() * sizeof(hiprandState);
    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevTabGM, sizeOctetGM);
	    Device::memclear(ptrDevTabGM, sizeOctetGM);
	    Device::malloc(&ptrTabDevGeneratorGM, sizeOctetGeneratorGM);
	    }

	Device::lastCudaError("Slice MM (end allocation)"); // temp debug, facultatif
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }

MonteCarlo::~MonteCarlo(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevTabGM);
	Device::free(ptrTabDevGeneratorGM);
	Device::lastCudaError("Slice MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MonteCarlo::run()
    {
    Device::lastCudaError("slice (before)"); // temp debug
    createGenerator<<<dg,db>>>(ptrTabDevGeneratorGM, Device::getDeviceId()); // assynchrone
    monteCarlo<<<dg,db, this->sizeOctetSM>>>(ptrDevTabGM,ptrTabDevGeneratorGM , this->nbThread);// assynchrone
    Device::lastCudaError("slice (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	Device::memcpyDToH(&n0, ptrDevTabGM, sizeOctetGM); // barriere synchronisation implicite
	}
    cout << "n0 : " << n0 << "\n";
    cout << "n : " << n << "\n";
    pi = 4.f * n0 / n;
    }

float MonteCarlo::getPI()
    {
    return pi;
    }

uint MonteCarlo::getN0()
    {
    return n0;
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
