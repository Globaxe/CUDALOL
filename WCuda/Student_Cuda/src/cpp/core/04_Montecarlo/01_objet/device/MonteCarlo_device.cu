#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <hiprand/hiprand_kernel.h>
#include "reductionADD.h"
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void monteCarlo(uint* ptrDevTabGM, hiprandState* ptrTabDevGeneratorGM, uint n);

__device__ float f(float x);

__device__ void useGenerator(hiprandState* tabDevGeneratorGM, uint n, uint tabSM[]);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void monteCarlo(uint* ptrDevTabGM, hiprandState* ptrTabDevGeneratorGM, uint n)
    {
    extern __shared__ uint tabSM[];
    useGenerator(ptrTabDevGeneratorGM, n, tabSM);
    __syncthreads();
    reductionADD<uint>(tabSM, ptrDevTabGM);
    }

__global__
void createGenerator(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    // Customisation du generator:
    // Proposition, au lecteur de faire mieux !
    // Contrainte : Doit etre différent d'un GPU à l'autre
    // Contrainte : Doit etre différent d’un thread à l’autre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }

__device__
void useGenerator(hiprandState* tabDevGeneratorGM, uint n, uint tabSM[])
    {
    const int TID = Indice1D::tid();

    const int TID_LOCAL = Indice1D::tidLocal();

// Global Memory -> Register (optimization)
    hiprandState localGenerator = tabDevGeneratorGM[TID];
    float xAlea;
    float yAlea;

    uint nx = 0;
    for (long i = 1; i <= n; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);

	if (yAlea <= f(xAlea))
	    {
	    nx++;
	    }

	}

    //Register -> Global Memory
    //Necessaire si on veut utiliser notre generator
    // - dans d’autre kernel
    // - avec d’autres nombres aleatoires !

    tabSM[TID_LOCAL] = nx;
    tabDevGeneratorGM[TID] = localGenerator;
    }

__device__ float f(float x)
    {
    return 1. / (1 + x * x);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

