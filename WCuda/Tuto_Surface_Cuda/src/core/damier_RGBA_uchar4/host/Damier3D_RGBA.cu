#include "hip/hip_runtime.h"
#include "Damier3D_RGBA.h"

#include <assert.h>

#include "Device.h"
#include "MathTools.h"
#include "StringTools.h"
#include "cudaTools.h"

using gpu::DomainMath3D;
using gpu::SurfaceStrip_RGBA_uchar4;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void damier3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n, float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Damier3D_RGBA::Damier3D_RGBA(const Grid& grid, int w, int h, const DomainMath3D& domaineMath, float dt, int n) :
	SurfaceStrip_RGBA_uchar4(w,h,domaineMath ), variateurAnimation(cpu::Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;
    this->dg = grid.dg;
    this->db = grid.db;

    // Tools
    this->title = "Damier3D_RGBA_uchar4";
    this->t = 0;
    }

Damier3D_RGBA::~Damier3D_RGBA()
    {
    // rien
    }

/*-------------------------*\
 |*	Surcharge	   *|
 \*------------------------*/

/**
 * Override
 * Call periodicly by the api
 */
void Damier3D_RGBA::fillVertex(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, unsigned int w, unsigned int h, const gpu::DomainMath3D& domaineMath)
    {
    damier3DCuda<<<dg,db>>>(ptrDevVerticesCoord,ptrDevVerticesColor,w,h,domaineMath,n,t);
    }

/**
 * Override
 * Call periodicly by the api
 */
void Damier3D_RGBA::animationStep(bool& isNeedUpdate)
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/**
 * Override
 * Call periodicly by the api
 */
void Damier3D_RGBA::paintPrimitives(Graphic2D& graphic2D)
    {
    graphic2D.setFont(TIMES_ROMAN_24);

    float r = 0.5f;
    float g = 0.5f;
    float b = 0.5f;

    graphic2D.setColorRGB(r, g, b);

    // TOP : Para Animation
	{
	string message = "t = " + StringTools::toString(t);
	graphic2D.drawTitleTop(message);
	}

    // BOTTOM : Title
	{
	graphic2D.drawTitleBottom(title);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

