#include "hip/hip_runtime.h"
#include "Damier3DMath_RGBA.h"
#include "Indice2D.h"
#include "IndiceTools_GPU.h"
#include "DomainMath3D_GPU.h"
#include "cudaTools.h"
#include "Device.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damier3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void damier3DCuda(float3* ptrDevVerticesCoord, uchar4* ptrDevVerticesColor, int w, int h, DomainMath3D domaineMath, int n, float t)
    {
    Damier3DMath_RGBA damierMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH = w * h;

    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w,h,&DX,&DY);

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX,DY,vertexI,vertexJ,&x,&y);

	damierMath.sommetXY(&sommet,x,y,t); // update sommet
	damierMath.colorZ(&color, sommet.z); // update color

	ptrDevVerticesColor[s] = color;
	ptrDevVerticesCoord[s] = sommet;

	s += NB_THREAD;
	}

    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

