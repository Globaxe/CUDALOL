#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "ImageVideoMath.h"

#include "IndiceTools_GPU.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void toGray(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void toGray(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    ImageVideoMath math = ImageVideoMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH=w*h;

    int pixelI;	// in [0,h[
    int pixelJ; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); 	// update (pixelI, pixelJ)

	math.colorIJ(&ptrDevPixels[s],pixelI, pixelJ, t); 	//update color  // ici pas besoin de i, j,t, optimisation possible

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

