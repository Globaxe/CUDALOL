#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "ImageVideo.h"

#include "OpencvTools_GPU.h"

using std::cout;
using std::cerr;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void toGray(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

ImageVideo::ImageVideo(const Grid& grid, uint w, uint h,  string nameVideo) :
Animable_I<uchar4>(grid, w, h, "ImageVideo_RGBA_uchar4"), nameVideo(nameVideo), capture(nameVideo), matRGBA(h, w, CV_8UC1)
    {
    // Tools
    this->t = 0;// protected dans Animable
    this->sizeImage = sizeof(uchar4) * w * h;

    //video
	{
	bool isOk = capture.start();
	if (!isOk)
	    {
	    cerr << "[ImageVideo] : failed to open : " << nameVideo << endl;
	    exit (EXIT_FAILURE);
	    }
	assert(capture.getW() == w && capture.getH() == h);
	animationStep(); // pour forcer la premiere capture et remplissage ptrTabPixelVideo
	}
    }

ImageVideo::~ImageVideo()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void ImageVideo::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::memcpyHToD(ptrDevPixels, ptrTabPixelVideo, sizeImage);

    Device::lastCudaError("image video rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    toGray<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("image video  rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void ImageVideo::animationStep()
    {
    t++;

    //video
	{
	Mat matBGR = capture.provideBGR();

	OpencvTools_GPU::switchRB(this->matRGBA, matBGR);
	this->ptrTabPixelVideo = OpencvTools_GPU::castToUchar4(matRGBA);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

