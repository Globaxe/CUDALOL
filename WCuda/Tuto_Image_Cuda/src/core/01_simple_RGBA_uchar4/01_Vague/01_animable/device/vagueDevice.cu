#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "VagueMath.h"

#include "IndiceTools_GPU.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    VagueMath vagueMath = VagueMath(w, h);

    const int WH=w*h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	vagueMath.colorIJ(&ptrDevPixels[s],i, j, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

