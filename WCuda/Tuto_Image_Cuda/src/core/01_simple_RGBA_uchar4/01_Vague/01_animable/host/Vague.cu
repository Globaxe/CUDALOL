#include "hip/hip_runtime.h"
#include "Vague.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vague(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Vague::Vague(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Vague_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

Vague::~Vague()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Vague::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("vague rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    vague<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("vague rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Vague::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

