#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DamierMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damier(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void damier(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t)
    {
    DamierMath damierMath = DamierMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    double x;
    double y;

    int i;// in [0,h[
    int j;// in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // update (i, j)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(i, j, &x, &y); //  (i,j) -> (x,y)

	damierMath.colorXY(&ptrDevPixels[s], x, y, t); // update ptrDevPixels[s]

	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

