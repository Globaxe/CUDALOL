#include "hip/hip_runtime.h"
#include "Damier.h"

#include <iostream>
#include <assert.h>

#include "Device.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damier(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Damier::Damier(const Grid& grid,uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Damier_Cuda_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

Damier::~Damier()
    {
   // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Damier::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("damier rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    damier<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);

    Device::lastCudaError("damier rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Damier::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

