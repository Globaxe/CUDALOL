#include "hip/hip_runtime.h"
#include "VagueGray.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vagueGray(uchar* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

VagueGray::VagueGray(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar>(grid, w, h, "Vague_Cuda_GRAY_uchar")
    {
    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    }

VagueGray::~VagueGray()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void VagueGray::process(uchar* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("vague gray uchar (before)"); // facultatif, for debug only, remove for release

    vagueGray<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("vague gray uchar (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void VagueGray::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

