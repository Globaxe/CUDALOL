#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "VagueGrayMath.h"
using namespace gpu;


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* ptrDevPixels, uint w, uint h, float t)
    {
    VagueGrayMath vagueGrayMath = VagueGrayMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH=w*h;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	vagueGrayMath.colorIJ(&ptrDevPixels[s],i, j, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

