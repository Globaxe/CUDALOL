#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DamierHueFloatMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t)
    {
    DamierHueFloatMath damierHueFloatMath = DamierHueFloatMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    double x;
    double y;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // update (i, j)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(i, j, &x, &y); //  (i,j) -> (x,y)

	damierHueFloatMath.colorXY(&ptrDevPixels[s], x, y, t); // update ptrDevPixels[s]

	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

