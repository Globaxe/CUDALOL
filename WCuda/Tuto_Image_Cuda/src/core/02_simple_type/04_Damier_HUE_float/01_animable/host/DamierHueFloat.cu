#include "hip/hip_runtime.h"
#include "DamierHueFloat.h"

#include <iostream>
#include <assert.h>

#include "Device.h"



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierHueFloat::DamierHueFloat(const Grid& grid,uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<float>(grid, w, h, "Damier_Cuda_Hue_float", domaineMath), variateurAnimation(Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierHueFloat::~DamierHueFloat()
    {
   // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierHueFloat::process(float* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("damier hue float (before)"); // facultatif, for debug only, remove for release

    damierHueFloat<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);

    Device::lastCudaError("damier hue float (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void DamierHueFloat::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

