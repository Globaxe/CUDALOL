#include "hip/hip_runtime.h"
#include "DamierRGBAFloat.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierRGBAFloat::DamierRGBAFloat(const Grid& grid,uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<float4>(grid, w, h, "Damier_Cuda_RGBA_float4", domaineMath), variateurAnimation(Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierRGBAFloat::~DamierRGBAFloat()
    {
   // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::process(float4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("damier rgba float4 (before)"); // facultatif, for debug only, remove for release

    damierRGBAFloat<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);

    Device::lastCudaError("damier rgba float4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

