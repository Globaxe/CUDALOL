#include "hip/hip_runtime.h"
#include "DamierHSBAFloat.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierHSBAFloat(float4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierHSBAFloat::DamierHSBAFloat(const Grid& grid, uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<float4>(grid, w, h, "Damier_Cuda_HSBA_float4", domaineMath), variateurAnimation(Interval<float>(0, 2 * PI), dt)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierHSBAFloat::~DamierHSBAFloat()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierHSBAFloat::process(float4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("damier hsba float4 (before)"); // facultatif, for debug only, remove for release

    damierHSBAFloat<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);

    Device::lastCudaError("damier hsba float4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void DamierHSBAFloat::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

